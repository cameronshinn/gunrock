// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * occupancy_api_timer.cu
 *
 * @brief Test program to check runtimes of CUDA occupancy API functions.
 */

#include <gunrock/oprtr/1D_oprtr/for.cuh>
#include <chrono>
#include <iostream>

using namespace std;

template <typename T>
void occupancyApiTimer(unsigned int *maxPotentialBlockSizeMicros, unsigned int *MaxActiveBlocksPerMultiprocessorMicros, T func)
{
    using namespace chrono;
    int blockSize;
    int minGridSize;
    int maxActiveBlocks;

    // Time hipOccupancyMaxPotentialBlockSize()
    steady_clock::time_point beginMpbs = steady_clock::now();
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, func, 0, 0);
    steady_clock::time_point endMpbs = steady_clock::now();
    *maxPotentialBlockSizeMicros = duration_cast<microseconds>(endMpbs - beginMpbs).count();

    // Time hipOccupancyMaxActiveBlocksPerMultiprocessor()
    steady_clock::time_point beginMabpm = steady_clock::now();
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, func, blockSize, 0);
    steady_clock::time_point endMabpm = steady_clock::now();
    *maxPotentialBlockSizeMicros = duration_cast<microseconds>(endMabpm - beginMabpm).count();
}

int main(void) {
    unsigned int mpbsMicros, mabpmMicros;
    auto dummyLambda = [] __host__ __device__ (int a) { return a; };
    occupancyApiTimer(&mpbsMicros, &mabpmMicros, gunrock::oprtr::For_Kernel<decltype(dummyLambda)>);
    cout << "hipOccupancyMaxPotentialBlockSize(): " << mpbsMicros / 1000000.0 << " seconds" << endl;
    cout << "hipOccupancyMaxActiveBlocksPerMultiprocessor(): " << mabpmMicros / 1000000.0 << " seconds" << endl;
}
