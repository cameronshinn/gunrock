// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file vn_app.cu
 *
 * @brief single-source shortest path (vn) application
 */

#include <gunrock/app/vn/vn_app.cuh>

namespace gunrock {
namespace app {
namespace vn {

hipError_t UseParameters(util::Parameters &parameters) {
  hipError_t retval = hipSuccess;
  GUARD_CU(UseParameters_app(parameters));
  GUARD_CU(UseParameters_problem(parameters));
  GUARD_CU(UseParameters_enactor(parameters));

  GUARD_CU(parameters.Use<std::string>(
      "src",
      util::REQUIRED_ARGUMENT | util::MULTI_VALUE | util::OPTIONAL_PARAMETER,
      "0",
      "<Vertex-IDs|random|largestdegree> The source vertices\n"
      "\tIf random, randomly select non-zero degree vertices;\n"
      "\tIf largestdegree, select vertices with largest degrees",
      __FILE__, __LINE__));

  GUARD_CU(parameters.Use<int>(
      "src-seed",
      util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::OPTIONAL_PARAMETER,
      util::PreDefinedValues<int>::InvalidValue,
      "seed to generate random sources", __FILE__, __LINE__));

  GUARD_CU(parameters.Use<int>(
      "srcs-per-run",
      util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::OPTIONAL_PARAMETER,
      util::PreDefinedValues<int>::InvalidValue,
      "number of source nodes per run", __FILE__, __LINE__));

  return retval;
}

} // namespace vn
} // namespace app
} // namespace gunrock

/*
 * @brief Simple interface take in graph as CSR format
 * @param[in]  num_nodes   Number of veritces in the input graph
 * @param[in]  num_edges   Number of edges in the input graph
 * @param[in]  row_offsets CSR-formatted graph input row offsets
 * @param[in]  col_indices CSR-formatted graph input column indices
 * @param[in]  edge_values CSR-formatted graph input edge weights
 * @param[in]  num_runs    Number of runs to perform vn
 * @param[in]  sources     Sources to begin traverse, one for each run
 * @param[in]  mark_preds  Whether to output predecessor info
 * @param[out] distances   Return shortest distance to source per vertex
 * @param[out] preds       Return predecessors of each vertex
 * \return     double      Return accumulated elapsed times for all runs
 */
double vn(const int num_nodes,
          const int num_edges,
          const int *row_offsets,
          const int *col_indices,
          const int *edge_values,
          int *sources,
          const bool mark_pred,
          int *distances,
          int *preds,
          const int num_runs,
          gunrock::util::Location allocated_on) {
  return vn<int, int, int, int>(num_nodes, num_edges, row_offsets, col_indices,
    edge_values, sources, mark_pred, distances, preds, num_runs, allocated_on);
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
